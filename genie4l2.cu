#include "genie4l2.h"
#include "genie/genie.h"
#include "genie/utility/Logger.h"

//let this cu files link all required cuda implementation

std::shared_ptr<genie::ExecutionPolicy> GenieBucketer::get_genie_policy()
{
    genie::utility::Logger::set_level(genie::utility::Logger::NONE);
    genie::Config config = genie::Config()
        .SetK(topk)
        .SetNumOfQueries(queryPerBatch)
        .SetGpuId(GPUID)
        .SetDim(sigDim);
    return genie::MakePolicy(config);
}


GenieBucketer::GenieBucketer(int topk, int queryPerBatch, int GPUID, int sigDim)
    :topk(topk), queryPerBatch(queryPerBatch), GPUID(GPUID), sigDim(sigDim)
{
    geniePolicy = get_genie_policy();
}


void GenieBucketer::build(const std::vector<std::vector<int> >& sigs)
{
    invTable = genie::BuildTable(geniePolicy, sigs);
}


namespace genie {
    namespace table {
        //the implementation of genie::table::InvertedTable::serialize
        template <class Archive>
        void InvertedTable::load(Archive &ar, const unsigned int )
        {
            ar >> inverted_index_;
            ar >> posting_list_;
            ar >> upperbounds_;
            ar >> lowerbounds_;
            ar >> num_of_rows_;
            ar >> num_of_dimensions_;
        }
        
        template <class Archive>
        void InvertedTable::save(Archive &ar, const unsigned int ) const
        {
            ar << inverted_index_;
            ar << posting_list_;
            ar << upperbounds_;
            ar << lowerbounds_;
            ar << num_of_rows_;
            ar << num_of_dimensions_;
        }
    }
};


std::vector<std::vector<int> > GenieBucketer::batch_query(const std::vector<std::vector<int> >& querySigs)
{
    auto genieQuery = genie::BuildQuery(geniePolicy, querySigs);
    auto genieResult = genie::Match(geniePolicy, invTable, genieQuery);
    
    //genieResult.first would be the idx and genieResult.second would be the count

    std::vector<std::vector<int> > ret;
    ret.resize(querySigs.size());
    for(int i=0;i<querySigs.size();i++){
        for(int j=0;j<topk;j++){
            int qidx = i*topk + j;
            // fmt::print("genieResult[{}]=({}, {})", qidx, genieResult.first[qidx][k], genieResult.second[qidx][k]);
            // printf("genieResult[%d]=(%d, %d)\n", qidx, genieResult.first[qidx], genieResult.second[qidx]);
            ret[i].push_back(genieResult.first[qidx]);
        }
    }
    return ret;
}



template<class Archive>
void GenieBucketer::serialize(Archive & ar, const unsigned int )
{
    ar & topk;
    ar & queryPerBatch;
    ar & GPUID;
    ar & sigDim;
    if(Archive::is_loading::value){
        geniePolicy = get_genie_policy();
        hipSetDevice(GPUID);
    }
    ar & invTable;
}

// Explicit template instantiation
template
void GenieBucketer::serialize(boost::archive::binary_iarchive & ar, const unsigned int version);
template
void GenieBucketer::serialize(boost::archive::binary_oarchive & ar, const unsigned int version);
template
void GenieBucketer::serialize(boost::archive::text_iarchive & ar, const unsigned int version);
template
void GenieBucketer::serialize(boost::archive::text_oarchive & ar, const unsigned int version);

