#include "genie4l2_dist.h"

int DistGenieBucketer::get_num_gpus()
{
    int devCount;
    hipGetDeviceCount(&devCount);
    return devCount;
}

DistGenieBucketer::DistGenieBucketer(int topk, int queryPerBatch, int sigDim)
    :topk(topk), queryPerBatch(queryPerBatch), sigDim(sigDim)
{
    numGPUs = get_num_gpus();

    bucketers.reserve(numGPUs);
    for(int i=0;i<numGPUs;i++){
        bucketers.emplace_back(topk, queryPerBatch, i, sigDim);
    }
}


void DistGenieBucketer::build(const std::vector<std::vector<std::vector<int> > >& sigs)
{
    assert(sigs.size()==numGPUs);

    //let each buckets build its own inv_table and so on.
    std::vector<std::thread> pools;
    // pools.clear();
    pools.reserve(numGPUs);

    std::vector<std::vector<std::vector<int> > > sigsForThreads;
    sigsForThreads.resize(numGPUs);

    //calc extents
    extents.resize(sigs.size()+1);
    extents[0] = 0;
    for(int i=0;i<sigs.size();i++){
        extents[i+1] = sigs[i].size();
    }
    //require c++17
    // std::exclusive_scan(extents.begin(), extents.end());
    for(int i=1;i<extents.size();i++) {
        extents[i] += extents[i-1];
    }

    for(int i=0;i<extents.size();i++){
        printf("extents[%d]=%d\n", i, extents[i]);
    }

    //build each bucketer
    for(int threadid=0;threadid<numGPUs;threadid++){
        pools.emplace_back([&sigs, threadid, this](){
            bucketers[threadid].build(sigs[threadid]);
        });
    }
    for(int threadid=0;threadid<numGPUs;threadid++){
        pools[threadid].join();
    }
}

std::vector<std::vector<int> > DistGenieBucketer::batch_query(const std::vector<std::vector<int> >& querySigs)
{
    std::vector<std::thread> pools;
    // pools.clear();
    pools.reserve(numGPUs);

    std::vector<std::vector<std::vector<int> > > candidates(numGPUs);

    //query each bucketer
    for(int threadid=0;threadid<numGPUs;threadid++){
        pools.emplace_back([&candidates, &querySigs, threadid, this](){
            auto ret = bucketers[threadid].batch_query(querySigs);
            candidates[threadid] = std::move(ret);
        });
    }
    for(int threadid=0;threadid<numGPUs;threadid++){
        pools[threadid].join();
    }

    std::vector<std::vector<int> > ret(querySigs.size());
    for(int i=0;i<ret.size();i++){
        for(int threadid=0;threadid<numGPUs;threadid++){
            for(int candidateFromBucketer:candidates[threadid][i]) {
                ret[i].push_back(candidateFromBucketer + extents[threadid]);
            }
        }
    }
    return ret;
}